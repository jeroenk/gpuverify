#include "hip/hip_runtime.h"
//pass
//--blockDim=[32,8] --gridDim=[20,80]

struct t {
  char *data;
  int step;
};

__global__ void computeVmapKernel(t map, char *p)
{
    __requires(map.step == 640);
    map.data = p;
    int u = threadIdx.x + blockIdx.x * blockDim.x;
    int v = threadIdx.y + blockIdx.y * blockDim.y;

    float z = 0;

    if (z != 0) {
      (map.data + v * map.step)[u] = u;
    } else {
      (map.data + v * map.step)[u] = u;
   }
}
