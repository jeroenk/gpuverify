//xfail:BOOGIE_ERROR
//--blockDim=2 --gridDim=1

#include <hip/hip_runtime.h>

__global__ void race_test (unsigned int* i, int* A)
{
  int tid = threadIdx.x;
  int j = atomicAdd(i,0);
  A[j] = tid;
}